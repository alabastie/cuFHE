/*
 * Test hipStreamSynchronize wrapper
 */

#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

int main() {
  SetSeed(); // set random seed

  PriKey pri_key; // private key
  PubKey pub_key; // public key
  Ptxt* pt = new Ptxt[3];
  Ctxt* ct = new Ctxt[3];
  Stream st1, st2;

  cout<< "------ Key Generation ------" <<endl;
  KeyGen(pub_key, pri_key);

  Initialize(pub_key); // essential for GPU computing

  Synchronize();

  for (int i = 0; i < 2; i++) {
    pt[i].message_ = rand() % Ptxt::kPtxtSpace;
    cout<< "i: "<<i<<"  message: " + pt[i].message_<<endl;
    Encrypt(ct[i], pt[i], pri_key);
  }

  And(ct[2], ct[1], ct[0], st1);

  StreamSynchronize(st1);

  Xor(ct[0], ct[1], ct[2], st2);

  StreamSynchronize(st2);

  Decrypt(pt[2], ct[0], pri_key);

  cout<<"out: " + pt[2].message_<<endl;

  if ((pt[0].message_ & pt[1].message_) ^ pt[1].message_ == pt[2].message_)
    cout<<"PASS"<<endl;
  else
    cout<<"FAIL"<<endl;

  return 0;
}